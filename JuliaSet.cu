#include "hip/hip_runtime.h"
//======================================================================================================================
//
// CUDA version of Julia set calculation
//
//! \file JuliaSet.cu
//! \author Sebastian Eibl <sebastian.eibl@fau.de>
//
//======================================================================================================================

#include "lodepng/lodepng.h"

#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <string>

typedef float real_t;

template <typename T>
real_t real_c(T val) { return static_cast<real_t>(val); }

#define CHECK(call)                                                                                                   \
{                                                                                                                     \
   const hipError_t error = (call);                                                                                  \
   if (error != hipSuccess)                                                                                          \
   {                                                                                                                  \
      std::cout << "Error: " << __FILE__ << ":"  << __LINE__ << std::endl;                                            \
      std::cout << "code: " << error << ", reason: " << hipGetErrorString(error) << std::endl;                       \
      exit(1);                                                                                                        \
   }                                                                                                                  \
}

__global__ void juliaGPU( const real_t cr,
                          const real_t ci,
                          const real_t originX,
                          const real_t originY,
                          const real_t width,
                          const real_t height,
                          const int numX,
                          const int numY,
                          const int ld,
                          int* level,
                          unsigned char* RGBpic)
{
   const int idx = blockIdx.x * blockDim.x + threadIdx.x;
   const int idy = blockIdx.y * blockDim.y + threadIdx.y;

   if ((idx < 2048) && (idy < 2048))
   {
      const int id = ld * idy + idx;
      const real_t x = originX + width  * ((real_t)(idx) / (real_t)(numX));
      const real_t y = originY + height * ((real_t)(idy) / (real_t)(numY));
      real_t z0r = x;
      real_t z0i = y;

      int i;
      for (i = 0; i < 100; ++i)
      {
         const real_t zrTemp = z0r;
         z0r = (z0r * z0r - z0i * z0i) + cr;
         z0i = 2 * zrTemp * z0i + ci;
         if ((z0r * z0r + z0i * z0i) > 100.0f)
         {
            break;
         }
      }
      level[id] = i;
   }
}

__global__ void findMinMax(const int n,
                           const int* level,
                           int* min,
                           int* max)
{
   __shared__ int sMin[512];
   __shared__ int sMax[512];
   const unsigned int tid = threadIdx.x;
   unsigned int i = blockIdx.x*(blockDim.x)+tid;
   unsigned int gridSize = blockDim.x * gridDim.x;
   sMin[tid] = level[i];
   sMax[tid] = level[i];

   i += gridSize;
   while (i<n)
   {
      if (sMin[tid] > level[i]) sMin[tid] = level[i];
      if (sMax[tid] < level[i]) sMax[tid] = level[i];
      i += gridSize;
   }

   __syncthreads();

   for (unsigned int s = blockDim.x/2; s>0; s>>=1)
   {
      if (tid < s)
      {
         if (sMin[tid] > sMin[tid + s]) sMin[tid] = sMin[tid + s];
         if (sMax[tid] < sMax[tid + s]) sMax[tid] = sMax[tid + s];
      }
      __syncthreads();
   }

   if (tid == 0)
   {
      min[blockIdx.x] = sMin[0];
      max[blockIdx.x] = sMax[0];
   }
}

__global__ void colorPicture( const int ld, const int min, const int max, const int* level, unsigned char* RGBpic) {
   const int idx = blockIdx.x * blockDim.x + threadIdx.x;
   const int idy = blockIdx.y * blockDim.y + threadIdx.y;

   if ((idx < 2048) && (idy < 2048))
   {
      const int id = ld * idy + idx;
      const real_t frac = static_cast<real_t>(level[id] - min) / static_cast<real_t>(max-min);
      RGBpic[id * 3 + 0] = static_cast<unsigned char> (static_cast<real_t>(255) * (1 - (1-frac) * (1-frac) * (1-frac) * (1-frac) * (1-frac) * (1-frac)) );
   }
}

int main(int argc, char **argv) {
   if (argc != 3)
   {
      std::cout << "./01_Julia blockDimX blockDimY" << std::endl;
      exit(EXIT_FAILURE);
   }
   const unsigned int size = 2048;
   const unsigned int blockDimX = atoi(argv[1]);
   const unsigned int blockDimY = atoi(argv[2]);

   // set up device
   hipSetDevice(0);

   std::vector<unsigned char> RGBpic;
   RGBpic.resize(size*size*3);
   for (int i = 0; i<RGBpic.size(); ++i)
   {
      RGBpic[i] = 0;
   }

   int* d_level;
   CHECK( hipMalloc((int**) &d_level, size * size * sizeof(int)) );
   std::vector<int> h_min(1024);
   int* d_min;
   CHECK( hipMalloc((int**) &d_min, 1024 * sizeof(int)) );
   std::vector<int> h_max(1024);
   int* d_max;
   CHECK( hipMalloc((int**) &d_max, 1024 * sizeof(int)) );
   unsigned char* d_pic;
   CHECK( hipMalloc((unsigned char**) &d_pic, RGBpic.size()) );
   CHECK( hipMemcpy(d_pic, &RGBpic[0], RGBpic.size(), hipMemcpyHostToDevice) );
   dim3 block( blockDimX, blockDimY, 1 );
   dim3 grid ( (2048+block.x-1)/block.x, (2048+block.y-1)/block.y, 1);
   juliaGPU<<< grid, block >>> (-0.0, 0.8, real_c(-2), real_c(-2), real_c(4), real_c(4), size, size, size, d_level, d_pic);
   CHECK( hipPeekAtLastError() );
   CHECK( hipDeviceSynchronize() );
   findMinMax<<< 1024, 512 >>> (size*size, d_level, d_min, d_max);
   CHECK( hipMemcpy(&h_min[0], d_min, h_min.size() * sizeof(int), hipMemcpyDeviceToHost) );
   CHECK( hipMemcpy(&h_max[0], d_max, h_max.size() * sizeof(int), hipMemcpyDeviceToHost) );
   std::cout << "min: " << *std::min_element(h_min.begin(), h_min.end()) << "\tmax: " << *std::max_element(h_max.begin(), h_max.end()) << std::endl;
   colorPicture<<< grid, block >>> (size, *std::min_element(h_min.begin(), h_min.end()), *std::max_element(h_max.begin(), h_max.end()), d_level, d_pic);
   CHECK( hipMemcpy(&RGBpic[0], d_pic, RGBpic.size(), hipMemcpyDeviceToHost) );
   lodepng::encode("julia.png", RGBpic, size, size, LCT_RGB);
   CHECK( hipFree(d_level) );
   CHECK( hipFree(d_pic) );
   return(0);
}
